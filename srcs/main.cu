#include "hip/hip_runtime.h"
#include "raytracer.cuh"
#include "camera.cuh"
#include "material.cuh"
#include <time.h>
#include <float.h>
#include <stdio.h>

__device__ vec3	ray_color(const ray &r, hittable **world, hiprandState *rand_state)
{
	ray				curr_ray;
	vec3			att;
	t_hit_record	rec;
	float			t;

	curr_ray = r;
	att = vec3(1, 1, 1);
	for (int i = 0; i < REFRACTION; ++i)
	{
		//if ((*world)->hit(curr_ray, 0.001f, FLT_MAX, rec))
		if (O_hit((hittable_list *)*world, curr_ray, 0.001f, FLT_MAX, rec))
		{
			ray		scattered;
			vec3	attenuation;
			if (rec.mat->scatter(curr_ray, rec, attenuation, scattered, rand_state))
			{
				att *= attenuation;
				curr_ray = scattered;
			}
			else
				return (vec3(0, 0, 0));
		}
		else
		{
			vec3 unit_direction = unit_vector(curr_ray.direction());
			t = 0.5f * (unit_direction.y() + 1.0f);
			return ((vec3(1,1,1)*(1.0f-t) + vec3(0.5,0.7,1.0)*t) * att);
		}
	}
	return (vec3(0, 0, 0));
}

__global__ void	rand_init(hiprandState *rand_state)
{
	int		x;
	int		y;
	int		i;

	x = blockDim.x * blockIdx.x + threadIdx.x;
	y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= W || y >= H)
		return ;
	i = W*y + x;

	hiprand_init(SEED, i, 0, &rand_state[i]);
}

__global__ void	render(vec3 *buf, camera **cam, hittable **world, hiprandState *rand_state)
{
	int			x;
	int			y;
	int			i;
	hiprandState	state;
	vec3		color;
	float		u;
	float		v;
	ray			r;

	#if SHARED
		__shared__ vec3	share_sam[BLOCK_H][BLOCK_W];
	#endif


	x = blockDim.x * blockIdx.x + threadIdx.x;
	y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= W || y >= H)
	{
		#if SHARED
			share_sam[threadIdx.y][threadIdx.x] = vec3(-1,-1,-1);
		#endif
		return ;
	}
	i = W*y + x;

	state = rand_state[i];
	color = vec3(0, 0, 0);
	for (int s = 0; s < SAMPLES; ++s)
	{
		u = float(x + hiprand_uniform(&state)) / float(W);
		v = float(y + hiprand_uniform(&state)) / float(H);
		r = O_get_ray(*cam, u, v, &state);
		color += ray_color(r, world, &state);
	}
	color /= float(SAMPLES);

	#if SHARED
		vec3	sam;
		float	counter;

		share_sam[threadIdx.y][threadIdx.x] = color;
		__syncthreads();
		sam = vec3(0,0,0);
		counter = 0;

		if (threadIdx.x+1 < blockDim.x)
		{
			sam += share_sam[threadIdx.y][threadIdx.x+1];
			++counter;
		}
		if (threadIdx.y+1 < blockDim.y)
		{
			sam += share_sam[threadIdx.y+1][threadIdx.x];
			++counter;
		}
		if (threadIdx.x-1 < blockDim.x)
		{
			sam += share_sam[threadIdx.y][threadIdx.x-1];
			++counter;
		}
		if (threadIdx.y-1 < blockDim.y)
		{
			sam += share_sam[threadIdx.y-1][threadIdx.x];
			++counter;
		}

		color = (1-WEIGHT) * color + WEIGHT * sam / counter;
	#endif

	color[0] = std::sqrt(color[0]);
	color[1] = std::sqrt(color[1]);
	color[2] = std::sqrt(color[2]);
	buf[i] = color;
}



void	write_color(std::ostream &out, vec3 pixel)
{
	out << int(255.99 * pixel.r()) << ' '
		<< int(255.99 * pixel.g()) << ' '
		<< int(255.99 * pixel.b()) << '\n';
}

void	print(vec3 *buf)
{
	std::cout << "P3\n" << W << " " << H << "\n255\n";
	for (int y = H-1; y >= 0; --y)
	{
		for (int x = 0; x < W; ++x)
			write_color(std::cout, buf[W*y + x]);
	}
}


#define RND (hiprand_uniform(&local_rand_state))
__global__ void	create_world(hittable **d_list, hittable **d_world, camera **d_camera, hiprandState *rand_state)
{
	if (threadIdx.x != 0 || blockIdx.x != 0)
		return ;

	hiprandState local_rand_state = *rand_state;
	d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
	int i = 1;
	for(int a = -11; a < 11; ++a)
	{
		for(int b = -11; b < 11; ++b)
		{
			float choose_mat = RND;
			vec3 center(a+RND,0.2,b+RND);
			if(choose_mat < 0.8f)
				d_list[i++] = new sphere(center, 0.2, new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
			else if(choose_mat < 0.95f)
				d_list[i++] = new sphere(center, 0.2, new metal(
					vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
			else
				d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
		}
	}
	d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
	d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
	d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
	*rand_state = local_rand_state;
	*d_world  = new hittable_list(d_list, 22*22+1+3);

	vec3 lookfrom(13,2,3);
	vec3 lookat(0,0,0);
	float dist_to_focus = 10.0; (lookfrom-lookat).length();
	float aperture = 0.1;
	*d_camera = new camera(lookfrom, lookat, vec3(0,1,0), 30.0, ASPECT_RATIO, aperture, dist_to_focus);
}

__global__ void	free_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
	if (threadIdx.x != 0 || blockIdx.x != 0)
		return ;
	for (int i = 0; i < 22*22+1+3; ++i)
	{
		delete ((sphere *)d_list[i])->mat;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

int	main(void)
{
	//vec3			*h_buf;
	vec3			*d_buf;
	hittable_list	**d_list;
	hittable_list	**d_world;
	hiprandState		*d_rand_state;
	hiprandState		*d_rand_state2;
	camera			**d_camera;
	clock_t			start;
	clock_t			stop;

	std::cerr << "Rendering a " << W << "x" << H << " image with " << SAMPLES;
	std::cerr << " samples per pixel in " << BLOCK_W << "x" << BLOCK_H << " blocks.\n";

	CHECK(hipMallocManaged((void **)&d_buf, BSIZE));
	//h_buf = (vec3 *)malloc(BSIZE);
	//CHECK(hipMalloc((void **)&d_buf, BSIZE));
	//CHECK(hipMemcpy(d_buf, h_buf, BSIZE, hipMemcpyHostToDevice));

	CHECK(hipMalloc((void **)&d_rand_state, PIXELS * sizeof(hiprandState)));
	CHECK(hipMalloc((void **)&d_rand_state2, sizeof(hiprandState)));
	CHECK(hipMalloc((void **)&d_list, (22*22+1+3)*sizeof(hittable *)));
	CHECK(hipMalloc((void **)&d_world, sizeof(hittable *)));
	CHECK(hipMalloc((void **)&d_camera, sizeof(camera *)));
	rand_init<<<1, 1>>>(d_rand_state2);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	create_world<<<1,1>>>((hittable **)d_list, (hittable **)d_world, d_camera, d_rand_state2);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	dim3 blocks(W / BLOCK_W + 1, H / BLOCK_H + 1);
	dim3 threads(BLOCK_W, BLOCK_H);
	rand_init<<<blocks, threads>>>(d_rand_state);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	start = clock();
	render<<<blocks, threads>>>(d_buf, d_camera, (hittable **)d_world, d_rand_state);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	stop = clock();
	std::cerr << "Took: " << ((double)(stop - start)) / CLOCKS_PER_SEC << "\n";

	//CHECK(hipMemcpy(h_buf, d_buf, BSIZE, hipMemcpyDeviceToHost));

	print(d_buf);
	//print(h_buf);

	free_world<<<1,1>>>((hittable **)d_list, (hittable **)d_world, d_camera);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	CHECK(hipFree(d_camera));
	CHECK(hipFree(d_list));
	CHECK(hipFree(d_world));
	CHECK(hipFree(d_rand_state));
	CHECK(hipFree(d_rand_state2));
	CHECK(hipFree(d_buf));
	//free(h_buf);

	return (0);
}

