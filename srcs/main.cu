#include "hip/hip_runtime.h"
#include "raytracer.hpp"
#include "camera.hpp"
#include "material.hpp"
#include <time.h>
#include <float.h>


__device__ vec3	ray_color(const ray &r, hittable **world, hiprandState *rand_state)
{
	ray				curr_ray;
	vec3			att;
	t_hit_record	rec;
	float			t;

	curr_ray = r;
	att = vec3(1, 1, 1);
	for (int i = 0; i < REFRACTION; ++i)
	{
		if ((*world)->hit(curr_ray, 0.001f, FLT_MAX, rec))
		{
			ray		scattered;
			vec3	attenuation;
			if (rec.mat->scatter(curr_ray, rec, attenuation, scattered, rand_state))
			{
				att *= attenuation;
				curr_ray = scattered;
			}
			else
				return (vec3(0, 0, 0));
		}
		else
		{
			vec3 unit_direction = unit_vector(curr_ray.direction());
			t = 0.5f * (unit_direction.y() + 1.0f);
			return ((vec3(1,1,1)*(1.0f-t) + vec3(0.5,0.7,1.0)*t) * att);
		}
	}
	return (vec3(0, 0, 0));
}

__global__ void	render_init(hiprandState *rand_state)
{
	int		x;
	int		y;
	int		i;

	x = blockDim.x * blockIdx.x + threadIdx.x;
	y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= W || y >= H)
		return ;
	i = W*y + x;

	hiprand_init(SEED, i, 0, &rand_state[i]);
}

__global__ void	render(vec3 *buf, camera **cam, hittable **world, hiprandState *rand_state)
{
	int			x;
	int			y;
	int			i;
	hiprandState	state;
	vec3		color;
	float		u;
	float		v;
	ray			r;

	x = blockDim.x * blockIdx.x + threadIdx.x;
	y = blockDim.y * blockIdx.y + threadIdx.y;
	if (x >= W || y >= H)
		return ;
	i = W*y + x;

	state = rand_state[i];
	color = vec3(0, 0, 0);
	for (int s = 0; s < SAMPLES; ++s)
	{
		u = float(x + hiprand_uniform(&state)) / float(W);
		v = float(y + hiprand_uniform(&state)) / float(H);
		r = (*cam)->get_ray(u, v, &state);
		color += ray_color(r, world, &state);
	}
	rand_state[i] = state;
	color /= float(SAMPLES);
	color[0] = std::sqrt(color[0]);
	color[1] = std::sqrt(color[1]);
	color[2] = std::sqrt(color[2]);
	buf[i] = color;
}

void	write_color(std::ostream &out, vec3 pixel)
{
	out << int(255.99 * pixel.r()) << ' '
		<< int(255.99 * pixel.g()) << ' '
		<< int(255.99 * pixel.b()) << '\n';
}

void	print(vec3 *buf)
{
	std::cout << "P3\n" << W << " " << H << "\n255\n";
	for (int y = H-1; y >= 0; --y)
	{
		for (int x = 0; x < W; ++x)
			write_color(std::cout, buf[W*y + x]);
	}
}

/*
https://raytracing.github.io/v3/books/RayTracingInOneWeekend.html
https://github.com/RayTracing/raytracing.github.io/
https://developer.nvidia.com/blog/accelerated-ray-tracing-cuda/
https://github.com/rogerallen/raytracinginoneweekendincuda?tab=readme-ov-file
*/


__global__ void	create_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
	if (threadIdx.x != 0 || blockIdx.x != 0)
		return ;
	d_list[0] = new sphere(vec3(0,0,-1), 0.5, new lambertian(vec3(0.1, 0.2, 0.5)));
	d_list[1] = new sphere(vec3(0,-100.5,-1), 100, new lambertian(vec3(0.8, 0.8, 0.0)));
	d_list[2] = new sphere(vec3(1,0,-1), 0.5, new metal(vec3(0.8, 0.6, 0.2), 0.0));
	d_list[3] = new sphere(vec3(-1,0,-1), 0.5, new dielectric(1.5));
	d_list[4] = new sphere(vec3(-1,0,-1), -0.45, new dielectric(1.5));
	*d_world = new hittable_list(d_list, 5);
	vec3	lookfrom(3, 3, 2);
	vec3	lookat(0, 0, -1);
	float	dist_to_focus = (lookfrom - lookat).length();
	float	aperture = 2.0f;
	*d_camera = new camera(lookfrom, lookat, vec3(0,1,0), 20.0, ASPECT_RATIO, aperture, dist_to_focus);
}

__global__ void	free_world(hittable **d_list, hittable **d_world, camera **d_camera)
{
	if (threadIdx.x != 0 || blockIdx.x != 0)
		return ;
	for (int i = 0; i < 5; ++i)
	{
		delete ((sphere *)d_list[i])->mat;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

int	main(void)
{
	vec3			*buf;
	hittable_list	**d_list;
	hittable_list	**d_world;
	hiprandState		*d_rand_state;
	camera			**d_camera;
	clock_t			start;
	clock_t			stop;

	std::cerr << "Rendering a " << W << "x" << H << " image with " << SAMPLES;
	std::cerr << " samples per pixel in " << BLOCK_W << "x" << BLOCK_H << " blocks.\n";

	CHECK(hipMallocManaged((void **)&buf, BSIZE));
	CHECK(hipMalloc((void **)&d_rand_state, PIXELS * sizeof(hiprandState)));
	CHECK(hipMalloc((void **)&d_list, 5*sizeof(hittable *)));
	CHECK(hipMalloc((void **)&d_world, sizeof(hittable *)));
	CHECK(hipMalloc((void **)&d_camera, sizeof(camera *)));
	create_world<<<1,1>>>((hittable **)d_list, (hittable **)d_world, d_camera);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());

	start = clock();
	dim3 blocks(W / BLOCK_W + 1, H / BLOCK_H + 1);
	dim3 threads(BLOCK_W, BLOCK_H);
	render_init<<<blocks, threads>>>(d_rand_state);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	render<<<blocks, threads>>>(buf, d_camera, (hittable **)d_world, d_rand_state);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	stop = clock();
	std::cerr << "Took: " << ((double)(stop - start)) / CLOCKS_PER_SEC << "\n";

	print(buf);

	free_world<<<1,1>>>((hittable **)d_list, (hittable **)d_world, d_camera);
	CHECK(hipGetLastError());
	CHECK(hipDeviceSynchronize());
	CHECK(hipFree(d_camera));
	CHECK(hipFree(d_list));
	CHECK(hipFree(d_world));
	CHECK(hipFree(d_rand_state));
	CHECK(hipFree(buf));

	return (0);
}

